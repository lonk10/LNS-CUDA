#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSparseToDense
#include <stdio.h>           
#include <stdlib.h>
#include "../include/lns.cuh"
#include "../include/init.cuh"




int computeMass(int ind, int *parts, int nodes_num, int *weights){
    int tot_mass = 0;
    for (int i = 0; i < nodes_num; i++){
        tot_mass += parts[ind*nodes_num+i]*weights[i];
    }
    return tot_mass;
}

int checkMass(int *parts, int *weights, int parts_num, int nodes_num, int max_mass){
    for (int i = 0; i < parts_num; i++){
        if (computeMass(i, parts, nodes_num, weights) > max_mass)
            return 0;
    }
    return 1;
}

int *computeNodeCost(int *parts, int *weights, int parts_num, int nodes_num, int *costs){
    for (int i = 0; i < parts_num; i++) costs[i] = 0;

    for (int i = 0; i < parts_num; i++){
        for (int j = 0; j < nodes_num; j++){
            costs[i] += parts[i*nodes_num + j] * weights[j];
        }
    }
    return costs;
}

void computeEdgeCost(int *parts, int part_id, CSR *row_rep, CSC *col_rep, int parts_num, int nodes_num, int edges_num, int *int_cost, int *ext_cost){
    int ind = 0;
    int start, end;
    int int_res = 0;
    int ext_res = 0;
    int node;
    for (int i = 0; i < nodes_num; i++){
        ind = part_id*nodes_num+i;
        if (parts[ind]){
            // out edges
            start = row_rep -> offsets[i];
            end = row_rep -> offsets[i+1];
            for (int j = start; j < end; j++){
                node = row_rep -> col_indexes[j];
                if (!parts[node]) {
                    int_res += row_rep -> values[j];
                } else {
                    ext_res += row_rep -> values[j];
                }
            }
            // in edges
            
            start = col_rep -> offsets[i];
            end = col_rep -> offsets[i+1];
            for (int j = start; j < end; j++){
                node = col_rep -> row_indexes[j];
                if (!parts[node]) {
                    int_res += col_rep -> values[j];
                } else {
                    ext_res += col_rep -> values[j];
                }
            }
        }
    }
    *int_cost = int_res;
    *ext_cost = ext_res;
}

void computeAllEdgeCost(int *parts, CSR *row_rep, CSC *col_rep, int parts_num, int nodes_num, int edges_num, int *int_costs, int *ext_costs){
    for (int i = 0; i < parts_num; i++){
        computeEdgeCost(parts, i, row_rep, col_rep, parts_num, nodes_num, edges_num, &int_costs[i], &ext_costs[i]);
    }
}

// Random functions

// Stores n*m/100 UNIQUE values in mask
void computeRandomMask(int * mask, int n, int m){
    int i = 0;
    int max = n*m/100;
    int *check = (int*) malloc(n*sizeof(int));
    int rand_node;
    for (int j = 0; j < n; j++){    // init
        check[j] = 0;
    }
    while (i < max){
        rand_node = rand() % n;
        if (check[rand_node] == 0){ // check if node has already been generated
            mask[i] = rand_node;
            check[rand_node] = 1;   // set node to generated
            i++;
        }
    }
    free(check);
}

void computeRandomAssignment(int * mask, int n, int m, int p){
    for (int i = 0; i < (n*m/100); i++){
        mask[i] = rand() % p;
    }
}

void removeFromCost(int *parts, int k, int n, int node, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    int res = 0;
    int start = csr_rep -> offsets[node];
    int end = csr_rep -> offsets[node+1];
    for (int z = start; z < end; z++){
        if (parts[k*n+z] == 0){ // only remove cost of edges going in/out of the partition
            ext_costs[k] -= csr_rep -> values[z];
        } else {
            int_costs[k] -= csr_rep -> values[z];
        }
    }
    start = csc_rep -> offsets[node];
    end = csc_rep -> offsets[node+1];
    for (int z = start; z < end; z++){
        if (parts[k*n+z] == 0){ // only add cost of edges going into the partition
            ext_costs[k] -= csc_rep -> values[z];
        } else {
            int_costs[k] -= csc_rep -> values[z];
        }
    }
}

int addToCost(int *parts, int k, int n, int node, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    int res = 0;
    int start = csr_rep -> offsets[node];
    int end = csr_rep -> offsets[node+1];
    for (int z = start; z < end; z++){
        if (parts[k*n+z] == 0){ // only add cost of edges going out of the partition
            ext_costs[k] += csr_rep -> values[z];
        } else {
            int_costs[k] += csr_rep -> values[z];
        }
    }
    start = csc_rep -> offsets[node];
    end = csc_rep -> offsets[node+1];
    for (int z = start; z < end; z++){
        if (parts[k*n+z] == 0){ // only add cost of edges going into the partition
            ext_costs[k] += csc_rep -> values[z];
        } else {
            int_costs[k] += csc_rep -> values[z];
        }
    }
    return res;
}

void destroy(int *parts, int k, int *destr_mask, int n, int m, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    int ind;
    /*
    for (int i = 0; i < k; i++){
        for (int j = 0; j < n; j++){
            ind = i*n+j;
            if (destr_mask[j] == 1 && parts[ind] == destr_mask[j]){
                parts[ind] = 0;
                //printf("destroyed node %d from part %d\n", j, i);
                edge_costs[k] -= removeFromCost(parts, k, n, j, edge_costs, csr_rep, csc_rep);
                node_costs[k] -= weights[j];
                //printf("updated costs\n");
            }
        }
    }*/
    int node;
    for (int i = 0; i < k; i++){
        for (int j = 0; j < (n*m/100); j++){
            node = destr_mask[j];
            ind = i*n+node;
            if (parts[ind] == 1){
                parts[ind] = 0;
                //printf("destroyed node %d from part %d\n", j, i);
                removeFromCost(parts, k, n, node, int_costs, ext_costs, csr_rep, csc_rep);
                //printf("updated costs\n");
            }
        }
    }
}

void repair(int *parts, int *destr_mask, int *asgn_mask, int n, int m, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    //int i = 0;
    int k;
    int node;
    for (int i = 0; i < (n*m/100); i++){
        k = asgn_mask[i];
        node = destr_mask[i];
        parts[k*n+node] = 1;
        //printf("added node %d to part %d\n", j, k);
        addToCost(parts, k, n, node, int_costs, ext_costs, csr_rep, csc_rep);
    }
}

float computeCost(int *int_costs, int *ext_costs, int k){
    float res = 0;
    float u = 0;
    for (int i = 0; i < k; i++){
        u = (float) 2*(int_costs[i]);
        printf("%f / (%f + %d = %f) = %f\n", u, u, ext_costs[i], (u+(float)ext_costs[i]), (u/ (u+(float)ext_costs[i])));
        res += u / (u + (float) ext_costs[i]); 
    }
    return res;
}

void lns(int *in_parts, int *weights, int parts_num, int nodes_num, int edges_num, int max_mass, int m, CSR *row_rep, CSC *col_rep){
    int *best = (int *) malloc(nodes_num*parts_num*sizeof(int));
    for (int i = 0; i < nodes_num*parts_num; i++){
        best[i] = in_parts[i];
    }
    //compute node costs
    int *int_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_int_cost = (int *)malloc(parts_num*sizeof(int));
    //computeNodeCost(best, weights, parts_num, nodes_num, node_cost);
    //compute edge costs
    int *ext_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_ext_cost = (int *)malloc(parts_num*sizeof(int));
    computeAllEdgeCost(best, row_rep, col_rep, parts_num, nodes_num, edges_num, int_cost, ext_cost);
    for (int i = 0; i < parts_num; i++){
        printf("init node cost %d \ninit edge cost %d\n", int_cost[i], ext_cost[i]);
    }
    float best_cost = computeCost(int_cost, ext_cost, parts_num);
    float new_cost;
    int destr_nodes = nodes_num*m/100;
    int *destr_mask = (int *)malloc(destr_nodes*sizeof(int));
    int *asgn_mask = (int *)malloc(destr_nodes*sizeof(int));
    int *temp = (int *) malloc(nodes_num*parts_num*sizeof(int));
    srand(time(NULL));

    printf("Initial cost is: %f\n", best_cost);

    for (int iter = 0; iter < MAX_ITER; iter++){
        //printf("Iteration %d start\n", iter);
        //reset values
        for (int i = 0; i < destr_nodes; i++){
            destr_mask[i] = 0;
        }
        memcpy(temp, in_parts, nodes_num*parts_num*sizeof(int));
        memcpy(temp_int_cost, int_cost, parts_num*sizeof(int));
        memcpy(temp_ext_cost, ext_cost, parts_num*sizeof(int));

        //printf("Destroy step %d\n", iter);
        //destroy step
        computeRandomMask(destr_mask, nodes_num, m);
        destroy(temp, parts_num, destr_mask, nodes_num, m, temp_int_cost, temp_ext_cost, row_rep, col_rep);

        //printf("Repair step %d\n", iter);
        //repair step
        computeRandomAssignment(asgn_mask, nodes_num, m, parts_num);
        repair(temp, destr_mask, asgn_mask, nodes_num, m, temp_int_cost, temp_ext_cost, row_rep, col_rep);

        //printf("Accept step %d\n", iter);
        //accept step
        if (checkMass(temp, weights, parts_num, nodes_num, max_mass)){
            new_cost = computeCost(temp_int_cost, temp_ext_cost, parts_num);
            if (new_cost > best_cost)
            printf("New best cost is: %f\n", new_cost);
                best_cost = new_cost;
                memcpy(best, temp, nodes_num*parts_num*sizeof(int));
        }
        //debug only
        //checkNodesPerPart(temp, parts_num, nodes_num);
        //checkPartsPerNode(temp, parts_num, nodes_num);
    }
    printf("Final cost is: %f\n", best_cost);
    printf("Partitions were:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", in_parts[i*nodes_num+j]);
        }
        printf("\n");
    }
    printf("Partitions are now:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", best[i*nodes_num+j]);
        }
        printf("\n");
    }
    //printf("snip:\n");
    free(destr_mask);
    //printf("snapp:\n");
    free(asgn_mask);
    //printf("snoop:\n");
}

int main(){
    int nodes_num, edges_num, parts_num;
    printf("Reading input...\n");
    FILE *in_file  = fopen("graph.txt", "r");
    char line[100];
    if (in_file == NULL){  
              printf("Error! Could not open file\n");
              exit(-1); // must include stdlib.h
            }
    // read number of nodes, edges and partitions
    fgets(line, 100, in_file);
    sscanf(line, "%d %d %d", &nodes_num, &edges_num, &parts_num);
    // init structures
    int *weights = (int *) malloc(nodes_num*sizeof(int));
    int *parts = (int *) malloc(nodes_num*sizeof(int));
    int *partitions = (int *) malloc(parts_num*nodes_num*sizeof(int));
    int *mat = (int *) malloc(nodes_num*nodes_num*sizeof(int));
    // read rest of the input
    readInput(in_file, partitions, weights, parts, nodes_num, edges_num, parts_num, mat);

    // setup csr representation
    int *h_csr_offsets = (int *) malloc((nodes_num + 1) * sizeof(int));
    int *h_csr_columns = (int *) malloc(edges_num * sizeof(int));
    int *h_csr_values = (int *) malloc(edges_num * sizeof(int));

    csrSetup(nodes_num, edges_num, mat, h_csr_offsets, h_csr_columns, h_csr_values);

    // setup csc representation
    int *h_csc_offsets = (int *) malloc((nodes_num + 1) * sizeof(int));
    int *h_csc_rows = (int *) malloc(edges_num * sizeof(int));
    int *h_csc_values = (int *) malloc(edges_num * sizeof(int));

    cscSetup(nodes_num, edges_num, mat, h_csc_offsets, h_csc_rows, h_csc_values);


    // generate csr rep
    // Device memory management

    //csrTest(h_csr_offsets, h_csr_columns, h_csr_values, nodes_num, edges_num);
    //cscTest(h_csc_offsets, h_csc_rows, h_csc_values, nodes_num, edges_num);

    CSR *row_rep = (CSR*) malloc(sizeof(CSR));
    row_rep -> offsets = h_csr_offsets;
    row_rep -> col_indexes = h_csr_columns;
    row_rep -> values = h_csr_values;

    CSC *col_rep = (CSC*) malloc(sizeof(CSC));
    col_rep -> offsets = h_csc_offsets;
    col_rep -> row_indexes = h_csc_rows;
    col_rep -> values = h_csc_values;

    lns(partitions, weights, parts_num, nodes_num, edges_num, MAX_MASS, DESTR_PERCENT, row_rep, col_rep);

    free(partitions);
    free(weights);
    free(parts);
    return 1;
}