#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSparseToDense
#include <stdio.h>           
#include <stdlib.h>
#include "../include/lns.cuh"
#include "../include/init.cuh"
#include "../include/util.cuh"

#define THREADS_PER_BLOCK 1024
#define GRIDS 10
#define BLOCKS_PER_ROW 1024

__device__ static void HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        //exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__inline__ __device__ int warpReduceSum(int val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// removes the edge cost of nodes in destr_mask
// from their corresponding partition in destr_parts

__global__ void removeNodes_v1(int* parts, int* nodes, int destr_nodes, int* int_costs, int* ext_costs, 
                            int* r_offset, int* r_indexes, int* r_values, int* c_offset, int* c_indexes, int* c_values,
                            int* removed_nodes) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < destr_nodes) {
        int node = nodes[ind];
        int k = parts[node];
        int start, end, edge_node, sum_i, sum_e;
        sum_i = 0;
        sum_e = 0;
        start = r_offset[node];
        end = r_offset[node+1];
        for (int i = start; i < end; i++){
            edge_node = r_indexes[i];
            if (parts[edge_node] == k){
                sum_i += (1 + !removed_nodes[edge_node]) * r_values[i];
            } else {
                sum_e += r_values[i];
            }
        }
        start = c_offset[node];
        end = c_offset[node+1];
        for (int i = start; i < end; i++){
            edge_node = c_indexes[i];
            if (parts[edge_node] == k){
                sum_i += (1 + !removed_nodes[edge_node]) * c_values[i];
            } else {
                sum_e += c_values[i];
            }
        }
        atomicSub(&int_costs[k], sum_i);
        atomicSub(&ext_costs[k], sum_e);
    }
}

__global__ void updatePartWeights(int* nodes, int* parts, int* out_i, int* out_e, int* costs_i, int* costs_e) {
    extern __shared__ int sdata[];

    sdata[threadIdx.x] = out_i[blockIdx.x * blockDim.x + threadIdx.x];
    sdata[threadIdx.x + blockDim.x] = out_e[blockIdx.x * blockDim.x + threadIdx.x];
    //printf("block %d of node %d int %d ext %d\n", blockIdx.x * blockDim.x + threadIdx.x, nodes[blockIdx.x], sdata[threadIdx.x], sdata[threadIdx.x+blockDim.x]);
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride && (threadIdx.x + stride) < blockDim.x) {
            sdata[threadIdx.x] += sdata[threadIdx.x + stride];
            sdata[threadIdx.x + blockDim.x] += sdata[threadIdx.x + blockDim.x + stride];
        }
        __syncthreads();
    } if (threadIdx.x < 32) {
        warpReduce(sdata, threadIdx.x);
        warpReduce(sdata, threadIdx.x + blockDim.x);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        int partition = parts[nodes[blockIdx.x]];
        //printf("removing node %d from parts %d, with weights %d and %d\n", nodes[blockIdx.x], partition, sdata[0], sdata[blockDim.x]);
        parts[nodes[blockIdx.x]] = -1;
        atomicSub(&costs_i[partition], sdata[0]);
        atomicSub(&costs_e[partition], sdata[blockDim.x]);
    }
}
// Given k partitions and n*m/100 threads per block
// each threads check if the destr_mask[threadIdx.x] node is present in its block's 
// partition and destroys it if necessary
// usage should be destroy<<k, n*m/100>>
// costs update should be handled by another function

__global__ void getPartitionPerDestrNode(int* parts, int* destr_mask, int* destr_parts, int destr_nodes, int n) {
    int partition = blockIdx.y;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < destr_nodes) {
        int node = destr_mask[tid];
        int ind = partition * n + node;
        if (parts[ind] == 1) {
            destr_parts[tid] = partition;
        }
    }
}
// Assigns n*m/100 nodes to random partions

__device__ float computePartCost(float u, float ext) {
    return 100 * (u / (u + ext));
}

__global__ void assignToParts_v1(int n, int* nodes, int destr_nodes, int* parts, int* int_costs, int* ext_costs, // util params
                                int *r_offset, int *r_indexes, int *r_values, int* c_offset, int* c_indexes, int* c_values, // graph
                                int *out_i, int *out_e) { // results

    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    if (ind < destr_nodes) {
        int node = nodes[ind];
        int start, end, res, edge_node, sum_i, sum_e;
        sum_i = 0;
        sum_e = 0;
        start = r_offset[node];
        end = r_offset[node+1];
        for (int i = start; i < end; i++){
            edge_node = r_indexes[i];
            if (parts[edge_node] == k){
                sum_i += r_values[i];
                if (parts[node] != k) sum_e -= res; // remove edge from the outer ones if part is not the og
            } else {
                sum_e += r_values[i];
            }
        }
        start = c_offset[node];
        end = c_offset[node+1];
        for (int i = start; i < end; i++){
            edge_node = c_indexes[i];
            if (parts[edge_node] == k){
                sum_i += c_values[i];
                if (parts[node] != k) sum_e -= res;
            } else {
                sum_e += c_values[i];
            }
        }
        out_i[ind*gridDim.y+k] = sum_i;
        out_e[ind*gridDim.y+k] = sum_e;
    }
}

__global__ void setToZero(int* arr, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) arr[tid] = 0;
}
__global__ void setRemovedNodes(int* nodes, int* arr, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) arr[nodes[tid]] = 1;
}


// removes nodes in destr_mask from 
void destroy_v1(int* parts, int* destr_mask, int destr_nodes, int k, int n, int* int_costs, int* ext_costs,
              int* r_offset, int* r_indexes, int* r_values, int* c_offset, int* c_indexes, int* c_values) {
    int* block_sums_i, * block_sums_e, * destr_parts, * removed_nodes;
    hipMalloc((void**)&block_sums_i, destr_nodes * BLOCKS_PER_ROW * sizeof(int));
    hipMalloc((void**)&block_sums_e, destr_nodes * BLOCKS_PER_ROW * sizeof(int));
    hipMalloc((void**)&destr_parts, destr_nodes * sizeof(int));
    // get partitions of destroyed nodes
    //getPartitionPerDestrNode << <dest_grid, THREADS_PER_BLOCK >> > (parts, destr_mask, destr_parts, destr_nodes, n);
    int sm_num; 
    hipDeviceGetAttribute(&sm_num, hipDeviceAttributeMultiprocessorCount, 0);
    int blockdim = min(1024, destr_nodes/sm_num);
    int gridx = destr_nodes/blockdim + (destr_nodes%blockdim > 0);
    dim3 grid_dim(gridx, 1, 1);
    dim3 block_dim(blockdim, 1, 1);
    hipMalloc((void**)&removed_nodes, n * sizeof(int));
    // set to zero removed_nodes
    setToZero << <n / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (removed_nodes, n);
    hipDeviceSynchronize();
    setRemovedNodes<<<destr_nodes / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> > (destr_mask, removed_nodes, destr_nodes);
    hipDeviceSynchronize();
    // remove nodes
    removeNodes_v1 << <grid_dim, block_dim, 2 * THREADS_PER_BLOCK * sizeof(int) >> > (parts, destr_mask, destr_nodes, int_costs, ext_costs, 
                                                                                    r_offset, r_indexes, r_values,
                                                                                    c_offset, c_indexes, c_values,
                                                                                    removed_nodes);
    hipDeviceSynchronize();
    //hipDeviceSynchronize(); // probably not needed
    hipFree(block_sums_i);
    hipFree(block_sums_e);
    hipFree(destr_parts);
    hipFree(removed_nodes);
}

// assigns node to partition with maximum score
// kernel is too small, only k threads and 1 block
// either find a fix or serialize this
// threads are 95+% instruction inactive
__global__ void assignToBestPart_v1(int k, float* results, int n, int* nodes, int* parts, int* int_costs, int* ext_costs, int* out_i, int* out_e) {
    int tid = threadIdx.x;
    extern __shared__ int sdata[];
    if (tid < k) { // reduction for finding index of max value in results
        sdata[tid] = tid; //initialize sdata to partition ids
    }
    __syncthreads();
    int nextTid;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            nextTid = sdata[tid + stride];
            if (results[blockIdx.x * k + sdata[tid]] < results[blockIdx.x * k + nextTid]) {
                sdata[tid] = nextTid;
            }

        }
        __syncthreads();
    }
    if (tid == 0) {
        int partition = sdata[0];
        parts[nodes[blockIdx.x]] = partition; // assign node to index sdata[0]
        atomicAdd(&int_costs[partition], 2 * out_i[blockIdx.x*k + partition]);
        atomicAdd(&ext_costs[partition], out_e[blockIdx.x * k + partition]);
    }

}

__global__ void gatherResults(int* out_i, int* out_e, int k, int* i_costs, int* e_costs, float* result, int n) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < n) {
        for (int i = 0; i < k; i++) {
            result[ind * k + i] = computePartCost(2 * out_i[ind * k + i] + i_costs[i], out_e[ind * k + i] + e_costs[i]) - computePartCost(i_costs[i], e_costs[i]);
        }
    }

}

void repair_v1(int* parts, int k, int* destr_mask, int n, int destr_nodes, int m, int* int_costs, int* ext_costs, 
             int* r_offset, int* r_indexes, int* r_values, int* c_offset, int* c_indexes, int* c_values) {
    //int i = 0;
    int node;
    float* d_result;
    int arr_size = k * destr_nodes;

    hipMalloc((void**)&d_result, arr_size * sizeof(float));
    float* result = (float*)malloc(arr_size * sizeof(float));
    int blocks = n / THREADS_PER_BLOCK + 1; // blocks/4 todo
    dim3 grid_dim(destr_nodes / THREADS_PER_BLOCK + 1, k, 1); // n/64 * k * m
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    int* out_e, * out_i;
    hipMalloc((void**)&out_e, arr_size * sizeof(int));
    hipMalloc((void**)&out_i, arr_size * sizeof(int));
    setToZero<<<arr_size/1024 + 1, 1024>>>(out_e, arr_size);
    setToZero<<<arr_size/1024 + 1, 1024>>>(out_i, arr_size);
    hipDeviceSynchronize();
    assignToParts_v1 << <grid_dim, block_dim, 4 * sizeof(int) >> > (n, destr_mask, destr_nodes, parts, int_costs, ext_costs, 
                                                                                      r_offset, r_indexes, r_values,
                                                                                      c_offset, c_indexes, c_values,
                                                                                      out_i, out_e);
    hipDeviceSynchronize();
    gatherResults << <destr_nodes/128 + 1, 128 >> > (out_i, out_e, k, int_costs, ext_costs, d_result, destr_nodes);
    hipDeviceSynchronize();
    assignToBestPart_v1 << <destr_nodes, k, 2 * k * sizeof(int) >> > (k, d_result, n, destr_mask, parts, int_costs, ext_costs, out_i, out_e);
    hipDeviceSynchronize();
    free(result);
    hipFree(d_result);
    hipFree(out_e);
    hipFree(out_i);
}

__global__ void resetMask(int* mask, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        mask[tid] = 0;
    }
}

void lns_v1(int* in_parts, int parts_num, int nodes_num, int edges_num, int max_mass, int m, CSR* row_rep, CSC* col_rep) {
    int* best = (int*)malloc(nodes_num * parts_num * sizeof(int));
    for (int i = 0; i < nodes_num; i++) {
        best[i] = in_parts[i];
    }
    //compute node costs
    int* d_temp_int_cost, * d_temp_ext_cost;

    int* int_cost = (int*)malloc(parts_num * sizeof(int));
    int* ext_cost = (int*)malloc(parts_num * sizeof(int));
    int* temp_int_cost = (int*)malloc(parts_num * sizeof(int));
    int* temp_ext_cost = (int*)malloc(parts_num * sizeof(int));
    hipMalloc((void**)&d_temp_int_cost, parts_num * sizeof(int));
    hipMalloc((void**)&d_temp_ext_cost, parts_num * sizeof(int));
    newComputeAllEdgeCost(best, row_rep, col_rep, parts_num, nodes_num, edges_num, int_cost, ext_cost);
    float best_cost = computeCost(int_cost, ext_cost, parts_num);
    float new_cost;
    int destr_nodes = nodes_num * m / 100;
    int* d_destr_mask, * temp;
    int* destr_mask = (int*)malloc(destr_nodes * sizeof(int));
    hipMalloc((void**)&d_destr_mask, destr_nodes * sizeof(int));
    hipMalloc((void**)&temp, nodes_num * sizeof(int));

    // copy CSR / CSC to device
    CSR* d_row_rep;
    CSC* d_col_rep;
    int* row_offsets, * col_offsets, * col_indexes, * row_indexes, * row_values, * col_values;
    printf("Allocation d_row, d_col\n");
    hipMalloc((void**)&d_row_rep, sizeof(CSR));
    hipMalloc((void**)&row_offsets, (nodes_num + 1) * sizeof(int));
    hipMalloc((void**)&col_indexes, edges_num * sizeof(int));
    hipMalloc((void**)&row_values, edges_num * sizeof(int));
    hipMalloc((void**)&d_col_rep, sizeof(CSC));
    hipMalloc((void**)&col_offsets, (nodes_num + 1) * sizeof(int));
    hipMalloc((void**)&row_indexes, edges_num * sizeof(int));
    hipMalloc((void**)&col_values, edges_num * sizeof(int));
    printf("Copying temps\n");
    hipMemcpy(&(d_row_rep->offsets), &row_offsets, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_row_rep->col_indexes), &col_indexes, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_row_rep->values), &row_values, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->offsets), &col_offsets, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->row_indexes), &row_indexes, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->values), &col_values, sizeof(int*), hipMemcpyHostToDevice);
    printf("Copying into temps\n");
    hipMemcpy(row_offsets, row_rep->offsets, (nodes_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_indexes, row_rep->col_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_values, row_rep->values, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_offsets, col_rep->offsets, (nodes_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_indexes, col_rep->row_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_values, col_rep->values, edges_num * sizeof(int), hipMemcpyHostToDevice);




    srand(time(NULL));

    printf("Initial cost is: %f\n", best_cost);

    for (int iter = 0; iter < MAX_ITER; iter++) {
        printf("*****\nIteration %d start\n*****\n", iter);
        //reset values
        for (int i = 0; i < destr_nodes; i++) {
            destr_mask[i] = 0;
        }
        hipMemcpy(temp, in_parts, nodes_num * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_int_cost, int_cost, parts_num * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_ext_cost, ext_cost, parts_num * sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num * sizeof(int), hipMemcpyDeviceToHost);


        //destroy step
        //printf("Random generation start\n");
        computeRandomMask(destr_mask, nodes_num, m);
        //printf("Random generation end\n");
        hipMemcpy(d_destr_mask, destr_mask, destr_nodes * sizeof(int), hipMemcpyHostToDevice);
        //printf("Destroy start\n");
        
        destroy_v1(temp, d_destr_mask, destr_nodes, parts_num, nodes_num, d_temp_int_cost, d_temp_ext_cost, 
                 row_offsets, col_indexes, row_values,
                 col_offsets, row_indexes, col_values);
        //printf("Destroy end\n");
        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num * sizeof(int), hipMemcpyDeviceToHost);
        printf("Cost after destroy %f \n", computeCost(temp_int_cost, temp_ext_cost, parts_num));


        //repair step
        //printf("Repair start\n");
        //repair(temp, parts_num, d_destr_mask, nodes_num, destr_nodes, m, d_temp_int_cost, d_temp_ext_cost, d_row_rep, d_col_rep);
        repair_v1(temp, parts_num, d_destr_mask, nodes_num, destr_nodes, m, d_temp_int_cost, d_temp_ext_cost, 
                row_offsets, col_indexes, row_values,
                col_offsets, row_indexes, col_values);
        //printf("Repair end\n");

        //accept step
        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num * sizeof(int), hipMemcpyDeviceToHost);


        if (checkMass(temp_int_cost, parts_num, max_mass)) {
            new_cost = computeCost(temp_int_cost, temp_ext_cost, parts_num);
            printf("New cost found is: %f\n", new_cost);
            if (new_cost > best_cost)
                //printf("New best cost is: %f\n", new_cost);
                best_cost = new_cost;
            hipMemcpy(best, temp, nodes_num * sizeof(int), hipMemcpyDeviceToHost);
        }
    }
    printf("Final cost is: %f\n", best_cost);

    //free
    free(best);
    free(int_cost);
    free(ext_cost);
    free(temp_int_cost);
    free(temp_ext_cost);
    free(destr_mask);
    //cudafree
    hipFree(d_temp_int_cost);
    hipFree(d_temp_ext_cost);
    hipFree(d_destr_mask);
    hipFree(temp);
    hipFree(row_offsets);
    hipFree(col_indexes);
    hipFree(row_values);
    hipFree(col_offsets);
    hipFree(row_indexes);
    hipFree(col_values);
    hipFree(d_row_rep);
    hipFree(d_col_rep);
}