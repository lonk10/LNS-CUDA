#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSparseToDense
#include <stdio.h>           
#include <stdlib.h>
#include "../include/lns.cuh"
#include "../include/init.cuh"
#include "../include/util.cuh"

#define THREADS_PER_BLOCK 256

// Given k partitions and n*m/100 threads per block
// each threads check if the destr_mask[threadIdx.x] node is present in its block's 
// partition and destroys it if necessary
// usage should be destroy<<k, n*m/100>>
// costs update should be handled by another function

__global__ void destroy(int *parts, int *destr_mask, int m){
    int tid = threadIdx.x;
    if (tid < m){
        int node = destr_mask[threadIdx.x];
        int ind = blockIdx.x * blockDim.x + node;
        if (parts[ind] == 1){
            parts[ind] = 0;
        }
    }
}

// Assigns n*m/100 nodes to random partions

__global__ void assignToParts(int n, int node, int *parts, float *result, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    //should be a parallel reduction here
    int k = blockIdx.x;
    int ind = threadIdx.x;
    extern __shared__ int sdata_i[];
    extern __shared__ int sdata_e[];

    int start_r = csr_rep -> offsets[node];
    int end_r = csr_rep -> offsets[node+1];
    int start_c = csc_rep -> offsets[node];
    int end_c = csc_rep -> offsets[node+1];

    int r_size = end_r - start_r;
    int c_size = end_c - start_c;
    int max_size = r_size > c_size ? r_size : c_size;
    /*
    if (ind == 0){
        hipMalloc( (void**)&sdata_i, max_size * sizeof(int));
        hipMalloc( (void**)&sdata_e, max_size * sizeof(int));
    }
    __syncthreads(); // wait for allocation*/
    // initialization
    sdata_i[ind] = 0;
    sdata_e[ind] = 0;
    __syncthreads();
    if (ind == 0)
        printf("Hello, thread %d of block %d init done\n", ind, k);
    __syncthreads();

    // gather values
    int edge_node;
    if (ind < r_size){
        edge_node = csr_rep -> col_indexes[start_r + ind];
        if (parts[k*n+edge_node]){
            sdata_i[ind] = csr_rep -> values[start_r + ind];
        } else {
            sdata_e[ind] = csr_rep -> values[start_r + ind];
        }
    }
    if (ind < c_size){
        edge_node = csc_rep -> row_indexes[start_c + ind];
        if (parts[k*n+edge_node]){
            sdata_i[ind] = csc_rep -> values[start_c + ind];
        } else {
            sdata_e[ind] = csc_rep -> values[start_c + ind];
        }
    }
    __syncthreads();
    if (ind == 0)
        printf("Hello, thread %d of block %d gathered values\n", ind, k);
    __syncthreads();
    

    // reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        if (ind < stride && (ind + stride) < max_size){
            sdata_i[ind] += sdata_i[ind + stride];
            sdata_e[ind] += sdata_e[ind + stride];
        }
        __syncthreads();
    }
    if (ind == 0){
        printf("Hello, thread %d of block %d reduction done\n", ind, k);
    }

    // store final result
    if (ind == 0){
        int mu_k = 2*(int_costs[k] + sdata_i[0]);
        result[k] = 100*((float) mu_k / (float)(mu_k + ext_costs[k] + sdata_e[0]));
        printf("mu_k: %d idata: %d edata: %d result: %f \n", mu_k, sdata_i[0], sdata_e[0], result[k]);
    }
    __syncthreads();
}

__global__ void assignToBestPart(int k, float *results, int n, int node, int *parts){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int sdata[];
    if (tid == 0){
        hipMalloc( (void**)&sdata, k * sizeof(int));
    }
    __syncthreads();
    if (tid < k){
        sdata[tid] = tid;
        __syncthreads();
        int nextTid;
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1){
            nextTid = sdata[tid + stride];
            if (tid < stride){
                if (results[tid] < results[tid + stride])
                    sdata[tid] = nextTid;
            }
            __syncthreads();
        }
        if (tid == 0){
            parts[sdata[0]*n+node] = 1;
            printf("Assigned node %d to part %d\n", node, sdata[0]);
        }
    }

}

void repair(int *parts, int k, int *destr_mask, int n, int m, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    //int i = 0;
    int node;
    float *d_result;
    hipMalloc( (void**)&d_result, k * sizeof(float));
    float *result = (float *) malloc(k*sizeof(float));
    int asgn;
    float temp_cost;
    for (int i = 0; i < (n*m/100); i++){
        //k = asgn_mask[i];
        node = destr_mask[i];
        assignToParts<<<k, THREADS_PER_BLOCK, 2 * n*n * sizeof(int)>>>(n, node, parts, d_result, int_costs, ext_costs, csr_rep, csc_rep);

        //debug stuff
        hipMemcpy(result, d_result, k*sizeof(float), hipMemcpyDeviceToHost);
        for (int z = 0; z < k; z++){
            printf("result[%d]: %d\n", z, result[z]);
        }
        hipDeviceSynchronize();
        assignToBestPart<<<k, THREADS_PER_BLOCK>>>(k, d_result, n, node, parts);
        hipDeviceSynchronize();
        /*
        hipMemcpy(d_result, result, k*sizeof(float), hipMemcpyDeviceToHost);
        asgn = 0;
        temp_cost = result[0];
        for (int j = 0; j < k; j++){
            if (result[j] > temp_cost){
                asgn = j;
                temp_cost = result[j];
                printf("new best result is result is %f\n", result[j]);
            }
        }
        
        //actual assign
        parts[asgn*n+node] = 1;
        printf("assigned node %d to part %d\n", node, asgn);*/
    }
    hipFree(result);
}

__global__ void resetMask(int *mask, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size){
        mask[tid] = 0;
    }
}

void lns_v1(int *in_parts, int *weights, int parts_num, int nodes_num, int edges_num, int max_mass, int m, CSR *row_rep, CSC *col_rep){
    int *best = (int *) malloc(nodes_num*parts_num*sizeof(int));
    for (int i = 0; i < nodes_num*parts_num; i++){
        best[i] = in_parts[i];
    }
    //compute node costs
    int *d_temp_int_cost, *d_temp_ext_cost;

    int *int_cost = (int *)malloc(parts_num*sizeof(int));
    int *ext_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_int_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_ext_cost = (int *)malloc(parts_num*sizeof(int));
    hipMalloc( (void**)&d_temp_int_cost, parts_num * sizeof(int));
    hipMalloc( (void**)&d_temp_ext_cost, parts_num * sizeof(int));
    computeAllEdgeCost(best, row_rep, col_rep, parts_num, nodes_num, edges_num, int_cost, ext_cost);
    for (int i = 0; i < parts_num; i++){
        printf("init node cost %d \ninit edge cost %d\n", int_cost[i], ext_cost[i]);
    }
    float best_cost = computeCost(int_cost, ext_cost, parts_num);
    float new_cost;
    int destr_nodes = nodes_num*m/100;
    int *d_destr_mask, *temp;
    int *destr_mask = (int *) malloc(destr_nodes * sizeof(int));
    hipMalloc( (void**)&d_destr_mask, destr_nodes * sizeof(int));
    hipMalloc( (void**)&temp, nodes_num * parts_num * sizeof(int));


    // copy CSR / CSC to device
    CSR *d_row_rep;
    CSC *d_col_rep;
    int *row_offsets, *col_offsets, *col_indexes, *row_indexes, *row_values, *col_values;
    printf("Allocation d_row, d_col\n");
    hipMalloc( (void**)&d_row_rep, sizeof(CSR));
    hipMalloc( (void**)&row_offsets, (nodes_num + 1) * sizeof(int));
    hipMalloc( (void**)&col_indexes, edges_num * sizeof(int));
    hipMalloc( (void**)&row_values, edges_num * sizeof(int));
    hipMalloc( (void**)&d_col_rep, sizeof(CSC));
    hipMalloc( (void**)&col_offsets, (nodes_num + 1) * sizeof(int));
    hipMalloc( (void**)&row_indexes, edges_num * sizeof(int));
    hipMalloc( (void**)&col_values, edges_num * sizeof(int));
    printf("Copying temps\n");
    hipMemcpy(&(d_row_rep->offsets), &row_offsets, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_row_rep->col_indexes), &col_indexes, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_row_rep->values), &row_values, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->offsets), &col_offsets, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->row_indexes), &row_indexes, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->values), &col_values, sizeof(int*), hipMemcpyHostToDevice);
    printf("Copying into temps\n");
    hipMemcpy(row_offsets, row_rep->offsets, (nodes_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_indexes, row_rep->col_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_values, row_rep->col_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_offsets, col_rep->offsets, (nodes_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_indexes, col_rep->row_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_values, col_rep->values, edges_num * sizeof(int), hipMemcpyHostToDevice);
    



    srand(time(NULL));

    printf("Initial cost is: %f\n", best_cost);

    for (int iter = 0; iter < MAX_ITER; iter++){
        //printf("Iteration %d start\n", iter);
        //reset values
        for (int i = 0; i < destr_nodes; i++){
            destr_mask[i] = 0;
        }
        //resetMask<<<parts_num, THREADS_PER_BLOCK>>>(destr_mask, destr_nodes);
        hipMemcpy(temp, in_parts, nodes_num*parts_num*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_int_cost, int_cost, parts_num*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_ext_cost, ext_cost, parts_num*sizeof(int), hipMemcpyHostToDevice);

        //printf("Destroy step %d\n", iter);
        //destroy step
        computeRandomMask(destr_mask, nodes_num, m);
        hipMemcpy(d_destr_mask, destr_mask, destr_nodes*sizeof(int), hipMemcpyHostToDevice);
        destroy<<<parts_num, THREADS_PER_BLOCK>>>(temp, d_destr_mask, m);
        hipDeviceSynchronize();

        //printf("Repair step %d\n", iter);
        //repair step
        //computeRandomAssignment(asgn_mask, nodes_num, m, parts_num);
        repair(temp, parts_num, destr_mask, nodes_num, m, d_temp_int_cost, d_temp_ext_cost, d_row_rep, d_col_rep);

        //printf("Accept step %d\n", iter);
        //accept step
        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);
        if (checkMass(temp_int_cost, parts_num, max_mass)){
            new_cost = computeCost(temp_int_cost, temp_ext_cost, parts_num);
            printf("New cost found is: %f\n", new_cost);
            if (new_cost > best_cost)
            printf("New best cost is: %f\n", new_cost);
                best_cost = new_cost;
                hipMemcpy(best, temp, nodes_num*parts_num*sizeof(int), hipMemcpyDeviceToHost);
        }
    }
    printf("Final cost is: %f\n", best_cost);
    printf("Partitions were:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", in_parts[i*nodes_num+j]);
        }
        printf("\n");
    }
    printf("Partitions are now:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", best[i*nodes_num+j]);
        }
        printf("\n");
    }
    //printf("snip:\n");
    hipFree(destr_mask);
    //printf("snapp:\n");
    //free(asgn_mask);
    //printf("snoop:\n");
}