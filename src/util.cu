#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/lns.cuh"
#include <stdlib.h>


int computeMass(int ind, int *parts, int nodes_num, int *weights){
    int tot_mass = 0;
    for (int i = 0; i < nodes_num; i++){
        tot_mass += parts[ind*nodes_num+i]*weights[i];
    }
    return tot_mass;
}

int checkMass(int *int_costs, int parts_num, int max_mass){
    for (int i = 0; i < parts_num; i++){
        if (int_costs[i] > max_mass){
            printf("mass check not passed, temp int cost[%d] is %d\n", i, int_costs[i]);
            return 0;
        }
        //printf("temp int cost[%d] is %d\n", i, int_costs[i]);
    }
    return 1;
}

float computeCost(int *int_costs, int *ext_costs, int k){
    float res = 0;
    float u = 0;
    for (int i = 0; i < k; i++){
        u = (float) (int_costs[i]);
        //printf("res %d:%f ", i, 100*(u/ (u+(float)ext_costs[i])));
        res += 100 * (u / (u + (float) ext_costs[i])); 
    }
    //printf("\n");
    return res;
}


void computeEdgeCost(int *parts, int part_id, CSR *row_rep, CSC *col_rep, int parts_num, int nodes_num, int edges_num, int *int_cost, int *ext_cost){
    int ind = 0;
    int start, end;
    int int_res = 0;
    int ext_res = 0;
    int node;
    for (int i = 0; i < nodes_num; i++){
        ind = part_id*nodes_num+i;
        if (parts[ind]){
            // out edges
            start = row_rep -> offsets[i];
            end = row_rep -> offsets[i+1];
            for (int j = start; j < end; j++){
                node = row_rep -> col_indexes[j];
                if (parts[part_id*nodes_num+node]) {
                    int_res += row_rep -> values[j];
                } else {
                    ext_res += row_rep -> values[j];
                }
            }
            // in edges
            
            start = col_rep -> offsets[i];
            end = col_rep -> offsets[i+1];
            for (int j = start; j < end; j++){
                node = col_rep -> row_indexes[j];
                if (parts[part_id*nodes_num+node]) {
                    int_res += col_rep -> values[j];
                } else {
                    ext_res += col_rep -> values[j];
                }
            }
        }
    }
    int_cost[part_id] = int_res;
    ext_cost[part_id] = ext_res;
}

void computeAllEdgeCost(int *parts, CSR *row_rep, CSC *col_rep, int parts_num, int nodes_num, int edges_num, int *int_costs, int *ext_costs){
    for (int i = 0; i < parts_num; i++){
        computeEdgeCost(parts, i, row_rep, col_rep, parts_num, nodes_num, edges_num, int_costs, ext_costs);
    }
}

void newComputeAllEdgeCost(int* parts, CSR* row_rep, CSC* col_rep, int parts_num, int nodes_num, int edges_num, int* int_costs, int* ext_costs) {
    for (int i = 0; i < parts_num; i++) {
        int_costs[i] = 0;
        ext_costs[i] = 0;
    }
    int start, end;
    int int_res = 0;
    int ext_res = 0;
    int partition;
    int node;
    for (int i = 0; i < nodes_num; i++) {
        int_res = 0;
        ext_res = 0;
        partition = parts[i];
        start = row_rep->offsets[i];
        end = row_rep->offsets[i + 1];
        for (int j = start; j < end; j++) {
            node = row_rep->col_indexes[j];
            if (parts[node] == parts[i]) {
                int_res += row_rep->values[j];
            }
            else {
                ext_res += row_rep->values[j];
            }
        }

        start = col_rep->offsets[i];
        end = col_rep->offsets[i + 1];
        for (int j = start; j < end; j++) {
            node = col_rep->row_indexes[j];
            if (parts[node] == parts[i]) {
                int_res += col_rep->values[j];
            }
            else {
                ext_res += col_rep->values[j];
            }
        }
        int_costs[partition] += int_res;
        ext_costs[partition] += ext_res;
    }
}

// Random functions


void computeRandomMask(int* mask, int n, int m) {
    int i = 0;
    int max = n * m / 100;
    //int *check = (int*) malloc(n*sizeof(int));
    int rand_node;
    unsigned char *is_used = (unsigned char *) malloc(n*sizeof(unsigned char)); /* flags */
    for (int z = 0; z < n; z++) {
        is_used[z] = 0;
    }

    int j = 0;
    int rn, rm;
    for (int i = n - max; i < n && j < max; i++){
        rand_node = rand() % (i+1);
        if (is_used[rand_node]) rand_node = i;
        mask[j++] = rand_node;
        //printf("generated %d\n", rand_node);
        is_used[rand_node] = 1;
        //printf("%d out of %d\n", i, max);
    }
    //printf("finished generation\n");
    free(is_used);
}

// Removes costs tied to node n in partition k
void removeFromCost(int *parts, int k, int n, int node, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    int res = 0;
    int start = csr_rep -> offsets[node];
    int end = csr_rep -> offsets[node+1];
    int edge_node;
    for (int z = start; z < end; z++){
        edge_node = csr_rep -> col_indexes[z];
        if (parts[k*n+edge_node] == 0){ // only remove cost of edges going in/out of the partition
            ext_costs[k] -= csr_rep -> values[z];
        } else {
            int_costs[k] -= csr_rep -> values[z];
        }
    }
    start = csc_rep -> offsets[node];
    end = csc_rep -> offsets[node+1];
    for (int z = start; z < end; z++){
        edge_node = csc_rep -> row_indexes[z];
        if (parts[k*n+edge_node] == 0){ // only add cost of edges going into the partition
            ext_costs[k] -= csc_rep -> values[z];
        } else {
            int_costs[k] -= csc_rep -> values[z];
        }
    }
}

// Adds costs tied to node n in partition k
int addToCost(int *parts, int k, int n, int node, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    int res = 0;
    int start = csr_rep -> offsets[node];
    int end = csr_rep -> offsets[node+1];
    int edge_node;
    for (int z = start; z < end; z++){
        edge_node = csr_rep -> col_indexes[z];
        if (parts[k*n+edge_node] == 0){ // only add cost of edges going out of the partition
            ext_costs[k] += csr_rep -> values[z];
        } else {
            int_costs[k] += csr_rep -> values[z];
        }
    }
    start = csc_rep -> offsets[node];
    end = csc_rep -> offsets[node+1];
    for (int z = start; z < end; z++){
        edge_node = csc_rep -> row_indexes[z];
        if (parts[k*n+edge_node] == 0){ // only add cost of edges going into the partition
            ext_costs[k] += csc_rep -> values[z];
        } else {
            int_costs[k] += csc_rep -> values[z];
        }
    }
    return res;
}