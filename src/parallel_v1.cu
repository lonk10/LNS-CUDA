#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSparseToDense
#include <stdio.h>           
#include <stdlib.h>
#include "../include/lns.cuh"
#include "../include/init.cuh"
#include "../include/util.cuh"

#define THREADS_PER_BLOCK 512
#define GRIDS 10
#define BLOCKS_PER_ROW 512

// removes the edge cost of nodes in destr_mask
// from their corresponding partition in destr_parts
__global__ void removeNodes(int *parts, int *destr_mask, int *destr_parts, int n, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep, int *block_sums_i, int *block_sums_e, int *removed_nodes){
    // indexes
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sdata[];
    int block_id = blockIdx.x;
    int k = blockIdx.y;

    // variables
    int partition = destr_parts[k];
    int node = destr_mask[k];

    // nodes in outgoing edges
    int start_r = csr_rep -> offsets[node];
    int end_r = csr_rep -> offsets[node+1];
    // nodes in incoming edges
    int start_c = csc_rep -> offsets[node];
    int end_c = csc_rep -> offsets[node+1];

    int r_size = end_r - start_r;
    int c_size = end_c - start_c;
    int max_size = r_size > c_size ? r_size : c_size;

    if (tid == 0){
        block_sums_i[k * gridDim.x + block_id] = 0;
        block_sums_e[k * gridDim.x + block_id] = 0;
    }
    if (ind == 0){
        removed_nodes[node] = 1;
    }
    __syncthreads();

    // go ahead only if there are enough nodes to handle
    if (ind < max_size){
        // initialize block sums
        sdata[ind] = 0;
        sdata[ind + blockDim.x] = 0;
        __syncthreads();

        // gather edge weights
        int edge_node;
        if (ind < r_size){
            edge_node = csr_rep -> col_indexes[start_r + ind];
            if (parts[partition*n+edge_node] == 1){
                if (removed_nodes[edge_node] == 1){ // this check is needed when multiple nodes from the same partition are removed, as to avoid removing the same edge twice
                    sdata[ind] = (csr_rep -> values[start_r + ind]);
                } else {
                    sdata[ind] = 2*(csr_rep -> values[start_r + ind]);
                }
            } else {
                sdata[ind+blockDim.x] = csr_rep -> values[start_r + ind];
            }
            
        }
        if (ind < c_size){
            edge_node = csc_rep -> row_indexes[start_c + ind];
            if (parts[partition*n+edge_node] == 1){
                if (removed_nodes[edge_node] == 1){
                    sdata[ind] = (csc_rep -> values[start_c + ind]);
                } else {
                    sdata[ind] = 2*(csc_rep -> values[start_c + ind]);
                }
            } else {
                sdata[ind+blockDim.x] = csc_rep -> values[start_c + ind];
            }
        }
        // reduction
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1){
            if (ind < stride && (ind + stride) < max_size){
                sdata[ind] += sdata[ind + stride];
                sdata[ind+blockDim.x] += sdata[ind + blockDim.x + stride];
            }
            __syncthreads();
        } 
    
        // store block reduction result
        // this is needed because a single can have max 1024 threads
        // so if a node is connected to more than 1024 other nodes, multiple blocks are needed to handle it
        // as shared memory is only intra-block, global memory is needed
        if (ind == 0){
            atomicAdd(&block_sums_i[k * gridDim.x + block_id], sdata[0]);
            atomicAdd(&block_sums_e[k * gridDim.x + block_id], sdata[blockDim.x]);
            //printf("Atomic adding %d and %d\n", sdata[0], sdata[blockDim.x]);
        }
        __syncthreads();

        // only the first thread of the grid row is going to update the costs
        // atomic operation needed as multiple nodes might have been removed
        // from the same partition, so there could be simultaneous writes 
        if (ind == 0 && block_id == 0){
            parts[partition*n+node] = 0; // remove node from partition

            int final_sum_i = 0;
            int final_sum_e = 0;
            for (int i = 0; i < gridDim.x; i++) {
                //printf("block sums %d out of %d: %d and %d\n", i, gridDim.x, block_sums_i[i], block_sums_e[i]);
                final_sum_i += block_sums_i[k * gridDim.x + i];
                final_sum_e += block_sums_e[k * gridDim.x + i];
            }

            //printf("Removing costs of node %d from part %d: %d and %d \n", node, partition, final_sum_i, final_sum_e);
            //printf("Costs are going from: %d and %d ", int_costs[partition], ext_costs[partition]);
            //update costs
            atomicSub(&int_costs[partition], final_sum_i); 
            atomicSub(&ext_costs[partition], final_sum_e);
            //printf("to: %d and %d\n", int_costs[partition], ext_costs[partition]);
        }
    }
}

// Given k partitions and n*m/100 threads per block
// each threads check if the destr_mask[threadIdx.x] node is present in its block's 
// partition and destroys it if necessary
// usage should be destroy<<k, n*m/100>>
// costs update should be handled by another function

__global__ void getPartitionPerDestrNode(int *parts, int *destr_mask, int *destr_parts, int destr_nodes){
    int tid = threadIdx.x;
    if (tid < destr_nodes){
        int node = destr_mask[tid];
        int ind = blockIdx.x * blockDim.x + node;
        if (parts[ind] == 1){
            //parts[ind] = 0;
            destr_parts[tid] = blockIdx.x;
            //printf("Thread %d of block %d destroyed node %d in partition %d\n", tid, blockIdx.x, node, blockIdx.x);
        }
    }
}
// Assigns n*m/100 nodes to random partions

__global__ void assignToParts(int n, int node, int *parts, float *result, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep, int *block_sums_i, int *block_sums_e){
    //should be a parallel reduction here
    int k = blockIdx.y;
    int block_id = blockIdx.x;
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int sdata[];

    int start_r = csr_rep -> offsets[node];
    int end_r = csr_rep -> offsets[node+1];
    int start_c = csc_rep -> offsets[node];
    int end_c = csc_rep -> offsets[node+1];

    int r_size = end_r - start_r;
    int c_size = end_c - start_c;
    int max_size = r_size > c_size ? r_size : c_size;

    // init of block sums
    // block_sums[k*gridDim.x...k*gridDim+block_id] is the reduction result of block block_id in row k
    if (threadIdx.x == 0){
        block_sums_i[k * gridDim.x + block_id] = 0;
        block_sums_e[k * gridDim.x + block_id] = 0;
    }
    __syncthreads();

    // sdata init
    if (ind < max_size) {
        sdata[ind] = 0;
        sdata[ind + blockDim.x] = 0;
    }

    // gather values
    int edge_node;
    //__syncthreads();
    if (ind < r_size){
        edge_node = csr_rep -> col_indexes[start_r + ind];
        if (parts[k*n+edge_node] == 1){
            sdata[ind] = csr_rep -> values[start_r + ind];
        } else {
            sdata[ind+blockDim.x] = csr_rep -> values[start_r + ind];
        }
        //printf("Outgoing edge node %d, part %d, sdata_i[%d] = %d, sdata_e[%d] = %d\n", edge_node, k, ind, sdata[ind], ind, sdata[ind+blockDim.x]);
        
    }
    if (ind < c_size){
        edge_node = csc_rep -> row_indexes[start_c + ind];
        if (parts[k*n+edge_node] == 1){
            sdata[ind] = csc_rep -> values[start_c + ind];
        } else {
            sdata[ind+blockDim.x] = csc_rep -> values[start_c + ind];
        }
        //printf("Incoming edge node %d, part %d, sdata_i[%d] = %d, sdata_e[%d] = %d\n", edge_node, k, ind, sdata[ind], ind, sdata[ind+blockDim.x]);
    }
    // reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        if (ind < stride && (ind + stride) < max_size){
            sdata[ind] += sdata[ind + stride];
            sdata[ind+blockDim.x] += sdata[ind + blockDim.x + stride];
        }
        __syncthreads();
    }

    if (ind == 0){
        atomicAdd(&block_sums_i[k * gridDim.x + block_id], sdata[0]);
        atomicAdd(&block_sums_e[k * gridDim.x + block_id], sdata[blockDim.x]);
    }

    // store final result
    if (ind == 0 && block_id == 0){
        int final_sum_i = 0;
        int final_sum_e = 0;
        for (int i = 0; i < gridDim.x; i++) {
            final_sum_i += block_sums_i[k * gridDim.x + i];
            final_sum_e += block_sums_e[k * gridDim.x + i];
        }
        int mu_k = (int_costs[k] + final_sum_i);
        result[k] = 100*((float) mu_k / (float)(mu_k + ext_costs[k] + final_sum_e));
        //printf("mu_k: %d idata: %d edata: %d result: %f \n", mu_k, final_sum_i, final_sum_e, result[k]);
    }
}


// removes nodes in destr_mask from 
// FIX REMOVEFROMCOSTS WHEN 0 EDGES
void destroy(int *parts, int *destr_mask, int destr_nodes, int k, int n, int *int_costs, int *ext_costs, CSR *row_rep, CSC *col_rep){
    int *block_sums_i, *block_sums_e, *destr_parts, *removed_nodes;
    hipMalloc( (void**)&block_sums_i, 256 * sizeof(int));
    hipMalloc( (void**)&block_sums_e, 256 * sizeof(int));
    hipMalloc( (void**)&destr_parts, destr_nodes * sizeof(int));
    getPartitionPerDestrNode<<<k, n>>>(parts, destr_mask, destr_parts, destr_nodes);
    hipDeviceSynchronize();
    dim3 gridDim(BLOCKS_PER_ROW, destr_nodes, 1);
    dim3 blockDim(THREADS_PER_BLOCK, 1, 1);
    hipMalloc( (void**)&removed_nodes, n * sizeof(int));
    removeNodes<<<gridDim, blockDim, 2 * THREADS_PER_BLOCK * sizeof(int)>>>(parts, destr_mask, destr_parts, n, int_costs, ext_costs, row_rep, col_rep, block_sums_i, block_sums_e, removed_nodes);
    hipDeviceSynchronize();
    hipFree(block_sums_i);
    hipFree(block_sums_e);
    hipFree(destr_parts);
    hipFree(removed_nodes);
}

// assigns node to partition with maximum score
__global__ void assignToBestPart(int k, float *results, int n, int node, int *parts, int *int_costs, int *ext_costs, int *block_sums_i, int *block_sums_e, int blocks){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int sdata[];
    if (tid < k){ // reduction for finding index of max value in results
        sdata[tid] = tid; //initialize sdata to partition ids
        //__syncthreads();
        int nextTid;
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1){
            nextTid = sdata[tid + stride];
            if (tid < stride){
                if (results[sdata[tid]] < results[nextTid])
                    sdata[tid] = nextTid;
            }
            __syncthreads();
        }
        if (tid == 0){
            int partition = sdata[0];
            parts[partition*n+node] = 1; // assign node to index sdata[0]
            int final_sum_i = 0;
            int final_sum_e = 0;
            for (int i = 0; i < blocks; i++) {
                final_sum_i += block_sums_i[partition * blocks + i];
                final_sum_e += block_sums_e[partition * blocks + i];
            }
            atomicAdd(&int_costs[partition], 2*final_sum_i);
            atomicAdd(&ext_costs[partition], final_sum_e);
            //printf("Assigned node %d to part %d\n", node, partition);
            //printf("Adding costs %d and %d\n", final_sum_i, final_sum_e);
        }
    }

}

void repair(int *parts, int k, int *destr_mask, int n, int edges_num, int m, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    //int i = 0;
    int node;
    float *d_result;
    hipMalloc( (void**)&d_result, k * sizeof(float));
    //float *result = (float *) malloc(k*sizeof(float));
    int asgn;
    float temp_cost;
    dim3 gridDim(n/THREADS_PER_BLOCK + 1, k, 1);
    dim3 blockDim(THREADS_PER_BLOCK, 1, 1);
    int *block_sums_e, *block_sums_i;
    hipMalloc( (void**)&block_sums_e, 5 * 256 * sizeof(int));
    hipMalloc( (void**)&block_sums_i, 5 * 256 * sizeof(int));
    for (int i = 0; i < (n*m/100); i++){
        node = destr_mask[i];
        assignToParts<<<gridDim, blockDim, 2 * THREADS_PER_BLOCK * sizeof(int)>>>(n, node, parts, d_result, int_costs, ext_costs, csr_rep, csc_rep, block_sums_i, block_sums_e);
        hipDeviceSynchronize();
        //debug stuff
        /*hipMemcpy(result, d_result, k*sizeof(float), hipMemcpyDeviceToHost);
        for (int z = 0; z < k; z++){
            //printf("result[%d]: %f\n", z, result[z]);
        }
        hipDeviceSynchronize();*/
        assignToBestPart<<<1, k, k*sizeof(int)>>>(k, d_result, n, node, parts, int_costs, ext_costs, block_sums_i, block_sums_e, n/THREADS_PER_BLOCK + 1);
        hipDeviceSynchronize();
    }
    //free(result);
    hipFree(d_result);
    hipFree(block_sums_e);
    hipFree(block_sums_i);
}

__global__ void resetMask(int *mask, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size){
        mask[tid] = 0;
    }
}

void lns_v1(int *in_parts, int parts_num, int nodes_num, int edges_num, int max_mass, int m, CSR *row_rep, CSC *col_rep){
    int *best = (int *) malloc(nodes_num*parts_num*sizeof(int));
    for (int i = 0; i < nodes_num*parts_num; i++){
        best[i] = in_parts[i];
    }
    //compute node costs
    int *d_temp_int_cost, *d_temp_ext_cost;

    int *int_cost = (int *)malloc(parts_num*sizeof(int));
    int *ext_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_int_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_ext_cost = (int *)malloc(parts_num*sizeof(int));
    hipMalloc( (void**)&d_temp_int_cost, parts_num * sizeof(int));
    hipMalloc( (void**)&d_temp_ext_cost, parts_num * sizeof(int));
    computeAllEdgeCost(best, row_rep, col_rep, parts_num, nodes_num, edges_num, int_cost, ext_cost);
    float best_cost = computeCost(int_cost, ext_cost, parts_num);
    float new_cost;
    int destr_nodes = nodes_num*m/100;
    int *d_destr_mask, *temp;
    int *destr_mask = (int *) malloc(destr_nodes * sizeof(int));
    hipMalloc( (void**)&d_destr_mask, destr_nodes * sizeof(int));
    hipMalloc( (void**)&temp, nodes_num * parts_num * sizeof(int));


    printf("Initial partitions:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", in_parts[i*nodes_num+j]);
        }
        printf("\n");
    }
    /*
    for (int i = 0; i < parts_num; i++){
        printf("init int_cost[%d] = %d\n", i, int_cost[i]);
        printf("init ext_cost[%d] = %d\n", i, ext_cost[i]);
    }*/

    // copy CSR / CSC to device
    CSR *d_row_rep;
    CSC *d_col_rep;
    int *row_offsets, *col_offsets, *col_indexes, *row_indexes, *row_values, *col_values;
    printf("Allocation d_row, d_col\n");
    hipMalloc( (void**)&d_row_rep, sizeof(CSR));
    hipMalloc( (void**)&row_offsets, (nodes_num + 1) * sizeof(int));
    hipMalloc( (void**)&col_indexes, edges_num * sizeof(int));
    hipMalloc( (void**)&row_values, edges_num * sizeof(int));
    hipMalloc( (void**)&d_col_rep, sizeof(CSC));
    hipMalloc( (void**)&col_offsets, (nodes_num + 1) * sizeof(int));
    hipMalloc( (void**)&row_indexes, edges_num * sizeof(int));
    hipMalloc( (void**)&col_values, edges_num * sizeof(int));
    printf("Copying temps\n");
    hipMemcpy(&(d_row_rep->offsets), &row_offsets, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_row_rep->col_indexes), &col_indexes, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_row_rep->values), &row_values, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->offsets), &col_offsets, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->row_indexes), &row_indexes, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_col_rep->values), &col_values, sizeof(int*), hipMemcpyHostToDevice);
    printf("Copying into temps\n");
    hipMemcpy(row_offsets, row_rep->offsets, (nodes_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_indexes, row_rep->col_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_values, row_rep->values, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_offsets, col_rep->offsets, (nodes_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_indexes, col_rep->row_indexes, edges_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_values, col_rep->values, edges_num * sizeof(int), hipMemcpyHostToDevice);
    



    srand(time(NULL));

    printf("Initial cost is: %f\n", best_cost);

    for (int iter = 0; iter < MAX_ITER; iter++){
        printf("*****\nIteration %d start\n*****\n", iter);
        //reset values
        for (int i = 0; i < destr_nodes; i++){
            destr_mask[i] = 0;
        }
        //resetMask<<<parts_num, THREADS_PER_BLOCK>>>(destr_mask, destr_nodes);
        hipMemcpy(temp, in_parts, nodes_num*parts_num*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_int_cost, int_cost, parts_num*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_ext_cost, ext_cost, parts_num*sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);
        
        /*
        for (int i = 0; i < parts_num; i++){
            printf("start temp_int_cost[%d] = %d\n", i, temp_int_cost[i]);
            printf("start temp_ext_cost[%d] = %d\n", i, temp_ext_cost[i]);
        }*/

        //printf("Destroy step %d\n", iter);
        //destroy step
        printf("Random generation start\n");
        computeRandomMask(destr_mask, nodes_num, m);
        printf("Random generation end\n");
        hipMemcpy(d_destr_mask, destr_mask, destr_nodes*sizeof(int), hipMemcpyHostToDevice);
        printf("Destroy start\n");
        destroy(temp, d_destr_mask, destr_nodes, parts_num, nodes_num, d_temp_int_cost, d_temp_ext_cost, d_row_rep, d_col_rep);
        printf("Destroy end\n");
        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);

        /*
        for (int i = 0; i < parts_num; i++){
            //printf("destr temp_int_cost[%d] = %d\n", i, temp_int_cost[i]);
            //printf("destr temp_ext_cost[%d] = %d\n", i, temp_ext_cost[i]);
        }*/

        //printf("Repair step %d\n", iter);
        //repair step
        //computeRandomAssignment(asgn_mask, nodes_num, m, parts_num);
        printf("Repair start\n");
        repair(temp, parts_num, destr_mask, nodes_num, edges_num, m, d_temp_int_cost, d_temp_ext_cost, d_row_rep, d_col_rep);
        printf("Repair end\n");

        //printf("Accept step %d\n", iter);
        //accept step
        hipMemcpy(temp_int_cost, d_temp_int_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(temp_ext_cost, d_temp_ext_cost, parts_num*sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < parts_num; i++){
            //printf("temp_int_cost[%d] = %d\n", i, temp_int_cost[i]);
            //printf("temp_ext_cost[%d] = %d\n", i, temp_ext_cost[i]);
        }

        if (checkMass(temp_int_cost, parts_num, max_mass)){
            new_cost = computeCost(temp_int_cost, temp_ext_cost, parts_num);
            //printf("New cost found is: %f\n", new_cost);
            if (new_cost > best_cost)
            //printf("New best cost is: %f\n", new_cost);
                best_cost = new_cost;
                hipMemcpy(best, temp, nodes_num*parts_num*sizeof(int), hipMemcpyDeviceToHost);
        }
    }
    printf("Final cost is: %f\n", best_cost);
    /*
    printf("Partitions were:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", in_parts[i*nodes_num+j]);
        }
        printf("\n");
    }
    printf("Partitions are now:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", best[i*nodes_num+j]);
        }
        printf("\n");
    }*/
    
    //free
    free(best);
    free(int_cost);
    free(ext_cost);
    free(temp_int_cost);
    free(temp_ext_cost);
    free(destr_mask);
    //cudafree
    hipFree(d_temp_int_cost);
    hipFree(d_temp_ext_cost);
    hipFree(d_destr_mask);
    hipFree(temp);
    hipFree(row_offsets);
    hipFree(col_indexes);
    hipFree(row_values);
    hipFree(col_offsets);
    hipFree(row_indexes);
    hipFree(col_values);
    hipFree(d_row_rep);
    hipFree(d_col_rep);
}