#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSparseToDense
#include <stdio.h>           
#include <stdlib.h>
#include "../include/lns.cuh"
#include "../include/init.cuh"
#include "../include/util.cuh"


void checkNodesPerPart(int *parts, int k, int n){
    int tot = 0;
    for (int i = 0; i < k*n; i++){
        tot += parts[i];
    }
    if (tot != n) printf("ERROR, counted %d nodes instead of %d.\n", tot, n);
    else printf("node check OK\n");
}

void checkPartsPerNode(int *parts, int k, int n){
    int res;
    for (int i = 0; i < n; i++){
        res = 0;
        for (int j = 0; j < k; j++){
            if (parts[j*n+i] == 1) res++;
        }
        if (res > 1) printf("Found node %d in multiple partitions\n", i);
    }
}

int *computeNodeCost(int *parts, int *weights, int parts_num, int nodes_num, int *costs){
    for (int i = 0; i < parts_num; i++) costs[i] = 0;

    for (int i = 0; i < parts_num; i++){
        for (int j = 0; j < nodes_num; j++){
            costs[i] += parts[i*nodes_num + j] * weights[j];
        }
    }
    return costs;
}



void computeRandomAssignment(int * mask, int n, int m, int p){
    for (int i = 0; i < (n*m/100); i++){
        mask[i] = rand() % p;
    }
}

void destroy(int *parts, int k, int *destr_mask, int n, int m, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    int ind;
    int node;
    for (int i = 0; i < k; i++){
        for (int j = 0; j < (n*m/100); j++){
            node = destr_mask[j];
            ind = i*n+node;
            if (parts[ind] == 1){
                parts[ind] = 0;
                //printf("destroyed node %d from part %d\n", node, i);
                removeFromCost(parts, i, n, node, int_costs, ext_costs, csr_rep, csc_rep);
                //printf("updated costs\n");
            }
        }
    }
}

void repair(int *parts, int *destr_mask, int n, int m, int parts_num, int *int_costs, int *ext_costs, CSR *csr_rep, CSC *csc_rep){
    //int i = 0;
    int k;
    int node;
    int best_k;
    float best_cost, temp_cost;
    int *temp_int_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_ext_cost = (int *)malloc(parts_num*sizeof(int));
    float old_cost = computeCost(int_costs, ext_costs, parts_num);
    
    for (int i = 0; i < (n*m/100); i++){
        node = destr_mask[i];
        best_cost = 0;
        best_k = 0;
        for (int j = 0; j < parts_num; j++){
            memcpy(temp_int_cost, int_costs, parts_num*sizeof(int));
            memcpy(temp_ext_cost, ext_costs, parts_num*sizeof(int));
            addToCost(parts, j, n, node, temp_int_cost, temp_ext_cost, csr_rep, csc_rep);
            temp_cost = computeCost(temp_int_cost, temp_ext_cost, parts_num) - old_cost;
            //printf("old cost: %f new cost: %f\n", best_cost, temp_cost);
            if (temp_cost > best_cost){
                best_cost = temp_cost;
                best_k = j;
            }
        }
        parts[best_k*n+node] = 1;
        addToCost(parts, best_k, n, node, int_costs, ext_costs, csr_rep, csc_rep);
    }
}


void lns_serial(int *in_parts, int parts_num, int nodes_num, int edges_num, int max_mass, int m, CSR *row_rep, CSC *col_rep){
    int *best = (int *) malloc(nodes_num*parts_num*sizeof(int));
    for (int i = 0; i < nodes_num*parts_num; i++){
        best[i] = in_parts[i];
    }
    //compute node costs
    int *int_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_int_cost = (int *)malloc(parts_num*sizeof(int));
    //computeNodeCost(best, weights, parts_num, nodes_num, node_cost);
    //compute edge costs
    int *ext_cost = (int *)malloc(parts_num*sizeof(int));
    int *temp_ext_cost = (int *)malloc(parts_num*sizeof(int));
    computeAllEdgeCost(best, row_rep, col_rep, parts_num, nodes_num, edges_num, int_cost, ext_cost);
    float best_cost = computeCost(int_cost, ext_cost, parts_num);
    float new_cost;
    int destr_nodes = nodes_num*m/100;
    int *destr_mask = (int *)malloc(destr_nodes*sizeof(int));
    int *temp = (int *) malloc(nodes_num*parts_num*sizeof(int));
    srand(time(NULL));

    printf("Initial cost is: %f\n", best_cost);

    for (int iter = 0; iter < MAX_ITER; iter++){
        printf("Iteration %d start\n", iter);
        //reset values
        for (int i = 0; i < destr_nodes; i++){
            destr_mask[i] = 0;
        }
        memcpy(temp, in_parts, nodes_num*parts_num*sizeof(int));
        memcpy(temp_int_cost, int_cost, parts_num*sizeof(int));
        memcpy(temp_ext_cost, ext_cost, parts_num*sizeof(int));

        //printf("Destroy step %d\n", iter);
        //destroy step
        computeRandomMask(destr_mask, nodes_num, m);
        destroy(temp, parts_num, destr_mask, nodes_num, m, temp_int_cost, temp_ext_cost, row_rep, col_rep);
        printf("cost after destroy: %f\n", computeCost(temp_int_cost, temp_ext_cost, parts_num));
        //printf("Repair step %d\n", iter);
        //repair step
        //computeRandomAssignment(asgn_mask, nodes_num, m, parts_num);
        repair(temp, destr_mask, nodes_num, m, parts_num, temp_int_cost, temp_ext_cost, row_rep, col_rep);

        //printf("Accept step %d\n", iter);
        //accept step
        if (checkMass(int_cost, parts_num, max_mass)){
            new_cost = computeCost(temp_int_cost, temp_ext_cost, parts_num);
            if (new_cost > best_cost)
            printf("New best cost is: %f\n", new_cost);
                best_cost = new_cost;
                memcpy(best, temp, nodes_num*parts_num*sizeof(int));
        }
        //debug only
        //checkNodesPerPart(temp, parts_num, nodes_num);
        //checkPartsPerNode(temp, parts_num, nodes_num);
    }
    printf("Final cost is: %f\n", best_cost);
    /*
    printf("Partitions were:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", in_parts[i*nodes_num+j]);
        }
        printf("\n");
    }
    printf("Partitions are now:\n");
    for (int i = 0; i < parts_num; i++){
        printf("Partition %d : ", i);
        for (int j = 0; j < nodes_num; j++){
            printf("%d", best[i*nodes_num+j]);
        }
        printf("\n");
    }*/
    //printf("snip:\n");
    free(destr_mask);
    //printf("snapp:\n");
    free(temp);
    free(int_cost);
    free(temp_int_cost);
    free(ext_cost);
    free(temp_ext_cost);
    //printf("snoop:\n");
}